
#include <hip/hip_runtime.h>
#include <string>
#include <vector>
#include <iostream>
#include <cstdlib>
#include <ctime>


#define N 25

using namespace std;
int tar;
int cnt;
void dfs(vector<int> numbers, int dep){
	if(numbers.size() == dep){
		int res=0;
		for(auto it = numbers.begin(); it < numbers.end(); it++){
			res += *it;
		}
		if(res == tar)
			cnt++;
		return ;
	}
	else{
		dfs(numbers,dep+1);
		numbers[dep] *= -1;
		dfs(numbers,dep+1);
	}
}


int solution(vector<int> numbers, int target) {
	int answer = 0;
	tar = target;

	dfs(numbers,0);  

	return cnt;
}

int main(){
	vector<int> numbers;
	int target;
	int random_var;

	srand(time(nullptr));

	for(int i=0; i<N; i++){
		random_var = rand()%51;
		//random_var = 1;
		numbers.push_back(random_var);
	}
	target = rand()%1001;
	//target = 3;


	int answer = solution(numbers,target);
	cout<<"###"<<endl;

	for(auto it = numbers.begin(); it<numbers.end(); it++){
		cout<<*it<<" ";
	}
	cout<<endl;

	cout<<"N : "<<N<<endl;
	cout<<"target number : "<<target<<endl;
	cout<<answer<<endl;
}
