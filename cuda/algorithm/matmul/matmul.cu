#include <iostream>
#include <hip/hip_runtime.h>

#define DATA_TYPE float
#define NX 1024*8 // A = NX * NY
#define NY 1024*32 // B = NY * NZ
#define NZ 1024
#define GPU_DEVICE 0

using namespace std;

__global__ void MatMul(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* Out){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	DATA_TYPE tmp=0;
	
	for(int i=0; i<NY; ++i){
		tmp += A[NY*idy+i] * B[i*NZ+idx];
	}
	Out[NX*idy + idx] = tmp;
	
}

void init_mat(DATA_TYPE* MAT, int size){
	for(int i=0; i<size; ++i){
		MAT[i]=1;
	}
}

void GPU_argv_init(){
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	cout<<"setting device "<< GPU_DEVICE << "with name" <<  deviceProp.name <<endl;
	hipSetDevice( GPU_DEVICE);
}

int main(){

	int size_A = NX*NY*sizeof(DATA_TYPE);
	int size_B = NY*NZ*sizeof(DATA_TYPE);
	int size_C = NX*NZ*sizeof(DATA_TYPE);

	DATA_TYPE* hA = (DATA_TYPE*)malloc(size_A);
	DATA_TYPE* hB = (DATA_TYPE*)malloc(size_B);
	DATA_TYPE* hC = (DATA_TYPE*)malloc(size_C); // result of matrix multiplication

	init_mat(hA, NX*NY);
	init_mat(hB, NY*NZ);

	DATA_TYPE* dA;
	DATA_TYPE* dB;
	DATA_TYPE* dC;

	GPU_argv_init();

	hipMalloc(&dA, size_A);
	hipMalloc(&dB, size_B);
	hipMalloc(&dC, size_C);

	hipMemcpy(dA, hA, size_A, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, size_B, hipMemcpyHostToDevice);

	dim3 block(32,8);
	dim3 grid(NY/block.x, 1);

	MatMul<<< block, grid >>>(dA,dB,dC);
	hipDeviceSynchronize();


	hipError_t err = hipGetLastError();
	if (err != hipSuccess) 
		cout<<"Error:"<< hipGetErrorString(err) <<endl;

	hipMemcpy(hC, dC, size_C, hipMemcpyDeviceToHost);

	for(int i=0; i<10; ++i){
		cout<<hC[i];
		if((i%128) == 127){
			cout<<endl;
		}
	}
	cout<<endl;

	hipFree(hA);
	hipFree(hB);
	hipFree(hC);

}

