#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define SIZE 1024*1024*4
#define HOST_ALLOC_FLAG hipHostMallocDefault // hipHostMalloc();
//#define HOST_ALLOC_FLAG hipHostMallocPortable // pinned memory by all CUDA context
//#define HOST_ALLOC_FLAG hipHostMallocMapped // Maps the allocation into the CUDA address space
//#define HOST_ALLOC_FLAG hipHostMallocWriteCombined // WC memory can be transferred across PCI Express bus, but cannot read efficiently by most CPUs. WC memory is a good option for buffers that will be written by CPU and read by the device
#define MALLOC_MANAGED_FLAG hipMemAttachGlobal // default
//#define MALLOC_MANAGED_FLAG hipMemAttachHost // Devices, has zero value for the device attribute hipDeviceAttributeConcurrentManagedAccess, can not access the memory.
#define HOST_REGISTER_FLAG hipHostRegisterDefault // mapped and protable
//#define HOST_REGISTER_FLAG hipHostRegisterPortable // pinned memory by all CUDA context
//#define HOST_REGISTER_FLAG hipHostRegisterMapped // maps the allocation into the CUDA addressspace
//#define HOST_REGISTER_FLAG hipHostRegisterIoMemory // pointing to some memory-mapped I/O space, belonging to a third-party PCIe device
#define ADVICE hipMemAdviseSetReadMostly // implies the data is mostly goint to be read and only occasionally written to. Any read accesses from any processor to this region will create a read-only copy
#define ADVICE hipMemAdviseUnsetReadMostly // undoes setReadMostly
//#define ADVICE cudaMemAdviseSetPrefferedLocation // sets the preferred location
//#define ADVICE cudaMemAdviseUnsetPrefferedLocation // unset the prefered location
//#define ADVICE hipMemAdviseSetAccessedBy // implies the data will be accessed by device
//#define ADVICE hipMemAdviseUnsetAccessedBy // undoes the SetAccessedBy



extern __shared__ float shared[]; // dynamic shared memory
								  // kernel<<< blocks, grid, shm_amount>> ...
__device__ __managed__ int var[2]; 

__global__ shm_foo(){

	__shared__ float shm_arr[10]; // static allocated shared memory

	float* shm_arr0 = (float*)shared; 
	int* shm_arr1 = (float*)&shm_arr0[128]; 
	var[0]=1;
	var[1]=2;

}

void GPU_argv_init(int dev_num){
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev_num);
	hipSetDevice( dev_num);
}

int main(){

	float* host_malloc;
	float* host_pinned;
	float* uvm_managed;
	float* pitched;
	float* dev_malloc;
	float* dev2_malloc;

	GPU_argv_init(0);

	/* allocate and register memory */
	host_malloc = (float*)malloc(SIZE);
	hipHostAlloc(host_pinned, SIZE, HOST_ALLOC_FLAG); //page-locked host memory
	hipMallocManaged(uvm_managed, SIZE, MALLOC_MANAGED_FLAG); //uvm memory
	hipHostRegister(host_malloc, SIZE, HOST_REGISTER_FLAG); //map or pin the host memory
	hipHostUnregister(host_malloc);	//unregisters a memory range
	hipMallocPitch(pitched, SIZE, 4096, 4096); // may pad the allocation to meet the alignment requirements
	hipMalloc(dev_malloc, SIZE); //malloc device global memory

	GPU_argv_init(1);
	hipMalloc(dev2_malloc,SIZE);
	GPU_argv_init(0);

	/* other CUDA memory runtime APIs */
	hipMemAdvise(uvm_managed, SIZE, ADVICE, DEVICE);
	hipMemset(dev_mallo, value=1, SIZE);
//	hipFuncSetAttribute(reinterpret_cast<const void*>(shm_foo), hipFuncAttributePreferredSharedMemoryCarveout, carveout); // split shared memory as 96/64/32/16/8/0 KB from unified shared/L1_cache memory.

	/* memory copy */
	hipMemcpy(dev_malloc, host_malloc, SIZE, hipMemcpyHostToDevice);
	hipMemcpy(host_malloc, dev_malloc, SIZE, hipMemcpyDeviceToHost);
	int stream=0;
	hipMemcpyAsync(dev_malloc, host_pinned, SIZE/4, hipMemcpyHostToDevice, ++stream);
	hipMemcpyPeer(dev2_malloc, 1, dev_malloc, 0, SIZE); // memory copy from GPU 0's memory to GPU 1's memory
	hipMemcpyPeerAsync(dev2_malloc, 1, dev_malloc, 0, SIZE/4, ++stream); // memory copy from GPU 0's memory to GPU 1's memory

	shm_foo<<<1,1>>>();
	cout << var[0] << var[1] <<endl;

	/* free */
	hipHostFree(host_pinned);
	hipFree(uvm_managed);
	hipFree(pitched);
	hipFree(dev_malloc);
	hipFree(dev2_malloc);

	return 0;
}
