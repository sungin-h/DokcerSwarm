#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__device__ volatile int X=1, Y=2;


__global__ kernel(){

	writeXY();
	readXY();
	//__threadfence_block(); // All writes to all memory and all reads from all memory made by the calling thread before the call to __threadfence_block()
	//__threadfence(); // no writes after the call to __threadfence();
	//__threadfence_system(); // acts as __threadfence_block() for all threads in the block are observed by all threads in the device, host threads, and all threads in peer devices

}
__device__ writeXY(){
	X=10;
	Y=20;
}
__device__ readXY(){
	int A=X;
	int B=Y;
}


int main(){


	return 0;
}
