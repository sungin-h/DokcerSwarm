#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void mykernel(int *addr){
	int min=1000, max=-1;
	int val = 10;
	atomicAdd(addr, 10);
	atomicSub(addr, 5);
	atomicExch(addr, 20);
	atomicMin(addr, min);
	atomicMax(addr, max);
	atomicInc(addr, val); // old >= val ? 0: ++old
	atomicDec(addr, val); // (old == 0) || (old > val) ? val : --old
	atomicCAS(addr, compare, val); // old==compare ? val : old
	atomicAnd(addr, val);
	atomicOr(addr, val);
	atommicXor(addr, val);
}


int main(){

	hipSetDevice(0);
	int *addr;
	hipMallocManaged(&addr, 4);
	*addr=0;

	mykernel<<<1,1>>>(addr);

	__sync_fetch_and_add(addr,10); // CPU atomic add operation

	return 0;

}
