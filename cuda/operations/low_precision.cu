#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <mma.h>

#define N 1024*1024*256

#define TB 256
#define ITER 1024*2
#define GRID 512


__global__ void double_precision(float* A, float* B, float* C){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i=0; i<ITER; ++i){
		C[i*TB*GRID + tid] = A[i*TB*GRID + tid] * B[i*TB*GRID + tid];
	}
}

__global__ void low_precision(int4*  A, int4* B, int4* C){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	for(int i=0; i<ITER; ++i){
		reinterpret_cast<int4*>(C)[i*TB*GRID + tid] = reinterpret_cast<int4*>(A)[i*TB*GRID + tid] + reinterpret_cast<int4*>(B)[i*TB*GRID + tid];
	}
}

int main(){
	float* host_A;
	float* host_B;
	float* host_C;
	float* dev_A;
	float* dev_B;
	float* dev_C;

	hipSetDevice(0);

	host_A = (float*)malloc(N*4);
	host_B = (float*)malloc(N*4);
	host_C = (float*)malloc(N*4);

	for(int i=0; i<N; ++i){
		host_A[i] = 1.0;
		host_B[i] = 2.0;
	}

	hipMalloc(&dev_A, N*4);
	hipMalloc(&dev_B, N*4);
	hipMalloc(&dev_C, N*4);

	hipMemcpy(dev_A, host_A, N*4, hipMemcpyHostToDevice);
	hipMemcpy(dev_B, host_B, N*4, hipMemcpyHostToDevice);

	double_precision<<<TB,GRID>>>(dev_A,dev_B,dev_C);
	hipDeviceSynchronize();
	
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
	free(host_A);
	free(host_B);
	free(host_C);

	return 0;
}
