#include "hip/hip_runtime.h"
#include <iostream>



#define ASYNC_FACTOR 2
#define SIZE 1024*1024*4
//#define STREAM_FLAG hipStreamDefault
//define STREAM_FLAG hipStreamNonBlocking

void GPU_argv_init(int dev_num){
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev_num);
	hipSetDevice( dev_num );

}
__global__ void mykernel(float* input, int len){
	
}

int main(){

	float* host_arr;
	float* host_pinned;

	float* dev_arr;

	GPU_argv_init(0);
	host_arr = (float*)malloc(SIZE*ASYNC_FACTOR);
	hipHostMalloc(&host_pinned, SIZE*ASYNC_FACTOR); //page-locked host memory
	hipHostRegister(host_arr, SIZE*ASYNC_FACTOR, hipHostRegisterPortable);
	hipMalloc(&dev_arr, SIZE*ASYNC_FACTOR);

	hipStream_t stream[ASYNC_FACTOR];
	for(int i=0; i<ASYNC_FACTOR; ++i){
		hipStreamCreate(&stream[i]);	
		//hipStreamCreateWithPriority(&stream[i],STREAM_FLAG ,i); //lower priority number represent high priority
	}

	for(int i=0; i<ASYNC_FACTOR; ++i){
		hipMemcpyAsync(dev_arr+i*SIZE, host_pinned, SIZE, hipMemcpyHostToDevice, stream[i]);
		mykernel<<<128,32,0,stream[i]>>>(dev_arr+i*SIZE, len);
		hipDeviceSynchronize();
		hipMemcpyAsync(host_arr+i*SIZE, dev_arr+i*SIZE, SIZE, hipMemcpyDeviceToHost, stream[i]);
	}

	if( hipStreamQuery(stream[0]) == hipSuccess){
		//stream[0] has been complete
	}

	for(int i=0; i<ASYNC_FACTOR; ++i){
		hipStreamSynchronize(stream[i]);
		hipStreamDestroy(stream[i]);
	}


	return 0;
}
